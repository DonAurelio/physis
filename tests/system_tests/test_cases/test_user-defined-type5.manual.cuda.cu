#include <stdio.h>
#include "hip/hip_runtime.h"


#define N 32
#define REAL float

#define OFFSET(x, y, z) ((x) + (y) * N + (z) * N * N)

 __global__ void kernel(REAL *g1, REAL *g2) {
   int x = threadIdx.x + blockIdx.x * blockDim.x;
   int y = threadIdx.y + blockIdx.y * blockDim.y;
   int z = threadIdx.z + blockIdx.z * blockDim.z;

   if (x == 0 || x == N-1 || y == 0 || y == N-1 ||
       z == 0 || z == N-1) return;
  
   float v = g1[OFFSET(x, y, z)] +
       g1[OFFSET(x+1, y, z)] + g1[OFFSET(x-1, y, z)] +
       g1[OFFSET(x, y+1, z)] + g1[OFFSET(x, y-1, z)] +
       g1[OFFSET(x, y, z-1)] + g1[OFFSET(x, y, z+1)];
   g2[OFFSET(x, y, z)] = v;
   return;
}

void dump(REAL *input) {
  int i;
  for (i = 0; i < N*N*N; ++i) {
    printf("%f\n", input[i]);
  }
}

#define halo_width (1)

int main(int argc, char *argv[]) {
  REAL *g1, *g1d;
  REAL *g2, *g2d;
  size_t nelms = N*N*N;
  g1 = (REAL *)malloc(sizeof(REAL) * nelms);
  g2 = (REAL *)malloc(sizeof(REAL) * nelms);  
  hipMalloc((void**)&g1d, sizeof(REAL) * nelms);
  hipMalloc((void**)&g2d, sizeof(REAL) * nelms);

  int i;
  for (i = 0; i < (int)nelms; i++) {
    g1[i] = i;
    g2[i] = 0;    
  }
    
  hipMemcpy(g1d, g1, sizeof(REAL) * nelms, hipMemcpyHostToDevice);
  hipMemcpy(g2d, g2, sizeof(REAL) * nelms, hipMemcpyHostToDevice);  
  
  dim3 block_dim(4, 4, 4);
  dim3 grid_dim(N/block_dim.x, N/block_dim.y, N/block_dim.z);

  kernel<<<grid_dim, block_dim>>>(g1d, g2d);
  hipMemcpy(g2, g2d, sizeof(REAL) * nelms, hipMemcpyDeviceToHost);

  dump(g2);

  hipDeviceReset();
  return 0;
}


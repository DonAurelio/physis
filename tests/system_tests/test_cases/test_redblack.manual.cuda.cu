#include <stdio.h>
#include "hip/hip_runtime.h"


#define N 32
#define REAL float

#define OFFSET(x, y, z) ((x) + (y) * N + (z) * N * N)

__global__ void kernel(REAL *g, int rb) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int z = threadIdx.z + blockIdx.z * blockDim.z;

  x = x * 2 + (y + z + rb) % 2;

  if (x == 0 || x == N-1 || y == 0 || y == N-1 ||
      z == 0 || z == N-1) return;
  
  float v = g[OFFSET(x, y, z)] +
      g[OFFSET(x+1, y, z)] + g[OFFSET(x-1, y, z)] +
      g[OFFSET(x, y+1, z)] + g[OFFSET(x, y-1, z)] +
      g[OFFSET(x, y, z-1)] + g[OFFSET(x, y, z+1)];
  g[OFFSET(x, y, z)] = v;
  return;
}

void dump(float *input) {
  int i;
  for (i = 0; i < N*N*N; ++i) {
    printf("%f\n", input[i]);
  }
}

#define halo_width (1)

int main(int argc, char *argv[]) {
  REAL *g, *gd;
  size_t nelms = N*N*N;
  g = (REAL *)malloc(sizeof(REAL) * nelms);
  hipMalloc((void**)&gd, sizeof(REAL) * nelms);

  int i;
  for (i = 0; i < (int)nelms; i++) {
    g[i] = i;
  }
    
  hipMemcpy(gd, g, sizeof(REAL) * nelms, hipMemcpyHostToDevice);
  
  dim3 block_dim(4, 4, 4);
  dim3 grid_dim(N/block_dim.x/2, N/block_dim.y, N/block_dim.z);

  kernel<<<grid_dim, block_dim>>>(gd, 0);
  kernel<<<grid_dim, block_dim>>>(gd, 1);  
  hipMemcpy(g, gd, sizeof(REAL) * nelms, hipMemcpyDeviceToHost);

  dump(g);

  hipDeviceReset();
  return 0;
}


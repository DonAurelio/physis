#include <stdio.h>
#include "hip/hip_runtime.h"


#define N 32
#define M (N+2)
#define REAL float

#define OFFSET1D(x) (x)
#define OFFSET2D(x, y) ((x) + (y) * M)
#define OFFSET3D(x, y, z) ((x) + (y) * N + (z) * N * M)


__global__ void kernel(REAL *g1, REAL *g2,
                       REAL *i, REAL *j) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int z = threadIdx.z + blockIdx.z * blockDim.z;

  if (x == 0 || x == N-1 || y == 0 || y == M-1 ||
      z == 0 || z == N-1) return;

  float v =
      g1[OFFSET3D(x, y, z)] +
      g1[OFFSET3D(x-1, y, z)] * i[OFFSET1D(x-1)] +
      g1[OFFSET3D(x+1, y, z)] * i[OFFSET1D(x+1)] +
      g1[OFFSET3D(x, y-1, z)] * j[OFFSET2D(y-1,z)] + 
      g1[OFFSET3D(x, y+1, z)] * j[OFFSET2D(y+1, z)] +
      g1[OFFSET3D(x, y, z-1)] * j[OFFSET2D(y,z-1)] + 
      g1[OFFSET3D(x, y, z+1)] * j[OFFSET2D(y,z+1)];
  g2[OFFSET3D(x, y, z)] = v;
  return;
}

void dump(float *input) {
  int i;
  for (i = 0; i < N*M*N; ++i) {
    printf("%f\n", input[i]);
  }
}

int main(int argc, char *argv[]) {
  REAL *g1, *g1d;
  REAL *g2d;
  REAL *ci, *cid;
  REAL *cj, *cjd;
  size_t nelms = N*M*N;
  g1 = (REAL *)malloc(sizeof(REAL) * nelms);
  ci = (REAL *)malloc(sizeof(REAL) * N);
  cj = (REAL *)malloc(sizeof(REAL) * M*N);  
  hipMalloc((void**)&g1d, sizeof(REAL) * nelms);
  hipMalloc((void**)&g2d, sizeof(REAL) * nelms);
  hipMalloc((void**)&cid, sizeof(REAL) * N);
  hipMalloc((void**)&cjd, sizeof(REAL) * M*N);

  int i;
  for (i = 0; i < (int)nelms; i++) {
    g1[i] = i;
  }
    
  hipMemcpy(g1d, g1, sizeof(REAL) * nelms, hipMemcpyHostToDevice);
  hipMemcpy(g2d, g1, sizeof(REAL) * nelms, hipMemcpyHostToDevice);

  for (i = 0; i < N; ++i) {
    ci[i] = 1 + (i%2); // 1 or 2    
  }

  for (i = 0; i < M*N; ++i) {
    cj[i] = 1 + (i%2); // 1 or 2
  }
  
  hipMemcpy(cid, ci, sizeof(REAL) * N, hipMemcpyHostToDevice);
  hipMemcpy(cjd, cj, sizeof(REAL) * (N+2) * N, hipMemcpyHostToDevice);  
  
  dim3 block_dim(16, 2, 1);
  dim3 grid_dim(N/block_dim.x, M/block_dim.y, N/block_dim.z);

  kernel<<<grid_dim, block_dim>>>(g1d, g2d, cid, cjd);
  hipMemcpy(g1, g2d, sizeof(REAL) * nelms, hipMemcpyDeviceToHost);

  dump(g1);

  hipDeviceReset();
  return 0;
}


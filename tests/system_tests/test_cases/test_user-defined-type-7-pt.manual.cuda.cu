#include <stdio.h>
#include <assert.h>
#include "hip/hip_runtime.h"


#define N 32

typedef struct {
  float p;
  float q;
} Point;

#define OFFSET(x, y, z) ((x) + (y) * N + (z) * N * N)

__global__ void kernel1(Point *g) {
   int x = threadIdx.x + blockIdx.x * blockDim.x;
   int y = threadIdx.y + blockIdx.y * blockDim.y;
   int z = threadIdx.z + blockIdx.z * blockDim.z;

   if (!(x > 0 && x < N-1 && y > 0 && y < N-1 && z > 0 && z < N-1)) {
     return;
   }

   float v = g[OFFSET(x, y, z)].p +
       g[OFFSET(x+1, y, z)].p +
       g[OFFSET(x-1, y, z)].p +
       g[OFFSET(x, y+1, z)].p +
       g[OFFSET(x, y-1, z)].p +
       g[OFFSET(x, y, z+1)].p +
       g[OFFSET(x, y, z-1)].p;
   g[OFFSET(x, y, z)].q = v;

   return;
}

void dump(Point *input) {
  int i;
  for (i = 0; i < N*N*N; ++i) {
    printf("%f %f\n", input[i].p, input[i].q);
  }
}

int main(int argc, char *argv[]) {
  Point *g_h, *g_d;
  size_t nelms = N*N*N;
  g_h = (Point *)malloc(sizeof(Point) * nelms);
  assert(hipSuccess ==
         hipMalloc((void**)&g_d, sizeof(Point) * nelms));

  int i;
  for (i = 0; i < (int)nelms; i++) {
    g_h[i].p = i;
    g_h[i].q = 0;
  }
    
  assert(hipSuccess ==
         hipMemcpy(g_d, g_h, sizeof(Point) * nelms,
                    hipMemcpyHostToDevice));
  dim3 block_dim(4, 4, 4);
  dim3 grid_dim(N/block_dim.x, N/block_dim.y, N/block_dim.z);

  kernel1<<<grid_dim, block_dim>>>(g_d);
  
  hipError_t e = hipGetLastError();
  if (e != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n",
            hipGetErrorString(e));
    exit(1);
  }

  assert(hipSuccess ==
         hipMemcpy(g_h, g_d, sizeof(Point) * nelms,
                    hipMemcpyDeviceToHost));

  dump(g_h);

  hipDeviceReset();
  return 0;
}


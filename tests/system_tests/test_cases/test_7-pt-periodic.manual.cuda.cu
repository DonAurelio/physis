#include <stdio.h>
#include "hip/hip_runtime.h"


#define N 32
#define REAL float

#define OFFSET(x, y, z) ((x) + (y) * N + (z) * N * N)

 __global__ void kernel(REAL *g1, REAL *g2) {
   int x = threadIdx.x + blockIdx.x * blockDim.x;
   int y = threadIdx.y + blockIdx.y * blockDim.y;
   int z = threadIdx.z + blockIdx.z * blockDim.z;

   int xp = ((x - 1) + N) % N;
   int xn = (x + 1) % N;
   int yp = ((y - 1) + N) % N;
   int yn = (y + 1) % N;
   int zp = ((z - 1) + N) % N;
   int zn = (z + 1) % N;
   float v =
       g1[OFFSET(x, y, z)] +
       g1[OFFSET(xn, y, z)] +
       g1[OFFSET(xp, y, z)] +
       g1[OFFSET(x, yn, z)] +
       g1[OFFSET(x, yp, z)] +
       g1[OFFSET(x, y, zn)] +
       g1[OFFSET(x, y, zp)];
   g2[OFFSET(x, y, z)] = v;
   return;
 }

void dump(float *input) {
  int i;
  for (i = 0; i < N*N*N; ++i) {
    printf("%f\n", input[i]);
  }
}

int main(int argc, char *argv[]) {
  REAL *g1, *g1d;
  REAL *g2d;
  size_t nelms = N*N*N;
  g1 = (REAL *)malloc(sizeof(REAL) * nelms);
  hipMalloc((void**)&g1d, sizeof(REAL) * nelms);
  hipMalloc((void**)&g2d, sizeof(REAL) * nelms);

  int i;
  for (i = 0; i < (int)nelms; i++) {
    g1[i] = i;
  }
    
  hipMemcpy(g1d, g1, sizeof(REAL) * nelms, hipMemcpyHostToDevice);
  hipMemcpy(g2d, g1, sizeof(REAL) * nelms, hipMemcpyHostToDevice);  
  
  dim3 block_dim(4, 4, 4);
  dim3 grid_dim(N/block_dim.x, N/block_dim.y, N/block_dim.z);

  kernel<<<grid_dim, block_dim>>>(g1d, g2d);
  hipMemcpy(g1, g2d, sizeof(REAL) * nelms, hipMemcpyDeviceToHost);

  dump(g1);

  hipDeviceReset();
  return 0;
}


#include "hip/hip_runtime.h"
#include "diffusion3d_cuda.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CUDA_SAFE_CALL(c)                       \
  do {                                          \
    assert(c == hipSuccess);                   \
  } while (0)

namespace diffusion3d {

__global__ void diffusion_kernel(REAL *f1, REAL *f2,
                                 int nx, int ny, int nz,
                                 REAL ce, REAL cw, REAL cn, REAL cs,
                                 REAL ct, REAL cb, REAL cc) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;  
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  int c = i + j * nx;
  int xy = nx * ny;
  for (int k = 0; k < nz; ++k) {
    int w = (i == 0)        ? c : c - 1;
    int e = (i == nx-1)     ? c : c + 1;
    int n = (j == 0)        ? c : c - nx;
    int s = (j == ny-1)     ? c : c + nx;
    int b = (k == 0)        ? c : c - xy;
    int t = (k == nz-1)     ? c : c + xy;
#if 1
    f2[c] = cc * f1[c] + cw * f1[w] + ce * f1[e] + cs * f1[s]
        + cn * f1[n] + cb * f1[b] + ct * f1[t];
#else
    // simulating the ordering of shared memory version
    REAL v = cc * f1[c];
    v += cw * f1[w];
    v += ce * f1[e];
    v += cs * f1[s];
    v += cn * f1[n];
    v += cb * f1[b] + ct * f1[t];
    f2[c] = v;
#endif    
    c += xy;
  }
  return;
}
#if 0
__global__ void diffusion_kernel_opt1(REAL *f1, REAL *f2,
                                      int nx, int ny, int nz,
                                      REAL ce, REAL cw, REAL cn, REAL cs,
                                      REAL ct, REAL cb, REAL cc) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;  
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  int c = i + j * nx;
  int xy = nx * ny;
  int w = (i == 0)        ? c : c - 1;
  int e = (i == nx-1)     ? c : c + 1;
  int n = (j == 0)        ? c : c - nx;
  int s = (j == ny-1)     ? c : c + nx;
  f2[c] = cc * f1[c] + cw * f1[w] + ce * f1[e] + cs * f1[s]
      + cn * f1[n] + cb * f1[c] + ct * f1[c+xy];
  c += xy;
  w += xy;
  e += xy;
  n += xy;
  s += xy;
  for (int k = 1; k < nz-1; ++k) {
#ifdef GST_BYPASS_L1
    float f = cc * f1[c] + cw * f1[w] + ce * f1[e] + cs * f1[s]
        + cn * f1[n] + cb * f1[c-xy] + ct * f1[c+xy];
    float *addr = &(f2[c]);
    // Since global stores in Femri bypasses L1 by default. The cg
    // option that forces L1 bypass does not affect performance.
    asm("st.global.cg.f32 [%0], %1;" :: "l"(addr), "f"(f));
#else    
    f2[c] = cc * f1[c] + cw * f1[w] + ce * f1[e] + cs * f1[s]
        + cn * f1[n] + cb * f1[c-xy] + ct * f1[c+xy];
#endif
    c += xy;
    w += xy;
    e += xy;
    n += xy;
    s += xy;
  }
  f2[c] = cc * f1[c] + cw * f1[w] + ce * f1[e] + cs * f1[s]
      + cn * f1[n] + cb * f1[c-xy] + ct * f1[c];
  return;
}
#else
// Use registers for z-direction points
__global__ void diffusion_kernel_opt1(REAL *f1, REAL *f2,
                                      int nx, int ny, int nz,
                                      REAL ce, REAL cw, REAL cn, REAL cs,
                                      REAL ct, REAL cb, REAL cc) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;  
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  int c = i + j * nx;
  int xy = nx * ny;
  int w = (i == 0)        ? c : c - 1;
  int e = (i == nx-1)     ? c : c + 1;
  int n = (j == 0)        ? c : c - nx;
  int s = (j == ny-1)     ? c : c + nx;
  REAL t1, t2, t3;
  t1 = t2 = f1[c];
  t3 = f1[c+xy];
  f2[c] = cc * t2 + cw * f1[w] + ce * f1[e] + cs * f1[s]
      + cn * f1[n] + cb * t1 + ct * t3;
  c += xy;
  w += xy;
  e += xy;
  n += xy;
  s += xy;
#pragma unroll 8
  for (int k = 1; k < nz-1; ++k) {
    t1 = t2;
    t2 = t3;
    t3 = f1[c+xy];
    f2[c] = cc * t2 + cw * f1[w] + ce * f1[e] + cs * f1[s]
        + cn * f1[n] + cb * t1 + ct * t3;
    c += xy;
    w += xy;
    e += xy;
    n += xy;
    s += xy;
  }
  t1 = t2;
  t2 = t3;
  f2[c] = cc * t2 + cw * f1[w] + ce * f1[e] + cs * f1[s]
      + cn * f1[n] + cb * t1 + ct * t3;
  return;
}
#endif

__global__ void diffusion_kernel_opt2(REAL *f1, REAL *f2,
                                      int nx, int ny, int nz,
                                      REAL ce, REAL cw, REAL cn, REAL cs,
                                      REAL ct, REAL cb, REAL cc) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;  
  int j = blockDim.y * blockIdx.y * 2 + threadIdx.y;
  int c = i + j * nx;
  int xy = nx * ny;
  int w = (i == 0)        ? c : c - 1;
  int e = (i == nx-1)     ? c : c + 1;
  int n = (j == 0)        ? c : c - nx;
  int s = (j == ny-1)     ? c : c + nx;
  f2[c] = cc * f1[c] + cw * f1[w] + ce * f1[e] + cs * f1[s]
      + cn * f1[n] + cb * f1[c] + ct * f1[c+xy];
  c += xy;
  w += xy;
  e += xy;
  n += xy;
  s += xy;
  for (int k = 1; k < nz-1; ++k) {
    f2[c] = cc * f1[c] + cw * f1[w] + ce * f1[e] + cs * f1[s]
        + cn * f1[n] + cb * f1[c-xy] + ct * f1[c+xy];
    c += xy;
    w += xy;
    e += xy;
    n += xy;
    s += xy;
  }
  f2[c] = cc * f1[c] + cw * f1[w] + ce * f1[e] + cs * f1[s]
      + cn * f1[n] + cb * f1[c-xy] + ct * f1[c];

  int offset = - xy * (nz-1) + nx * blockDim.y;
  c += offset;
  w += offset;
  e += offset;
  int j2 = blockDim.y * blockIdx.y * 2 + threadIdx.y + blockDim.y;
  n = (j2 == 0)        ? c : c - nx;
  s = (j2 == ny-1)     ? c : c + nx;
  f2[c] = cc * f1[c] + cw * f1[w] + ce * f1[e] + cs * f1[s]
      + cn * f1[n] + cb * f1[c] + ct * f1[c+xy];
  c += xy;
  w += xy;
  e += xy;
  n += xy;
  s += xy;
  for (int k = 1; k < nz-1; ++k) {
    f2[c] = cc * f1[c] + cw * f1[w] + ce * f1[e] + cs * f1[s]
        + cn * f1[n] + cb * f1[c-xy] + ct * f1[c+xy];
    c += xy;
    w += xy;
    e += xy;
    n += xy;
    s += xy;
  }
  f2[c] = cc * f1[c] + cw * f1[w] + ce * f1[e] + cs * f1[s]
      + cn * f1[n] + cb * f1[c-xy] + ct * f1[c];
  return;
}

#if 0
__global__ void diffusion_kernel_xy(REAL *f1, REAL *f2,
                                    int nx, int ny, int nz,
                                    REAL ce, REAL cw, REAL cn, REAL cs,
                                    REAL ct, REAL cb, REAL cc) {
  int bdimx = blockDim.x;
  int i = bdimx * blockIdx.x * 2 + threadIdx.x;  
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  int c = i + j * nx;
  int xy = nx * ny;
  for (int k = 0; k < nz; ++k) {
    int w = (i == 0)        ? c : c - 1;
    int e = (i == nx-1)     ? c : c + 1;
    int n = (j == 0)        ? c : c - nx;
    int s = (j == ny-1)     ? c : c + nx;
    int b = (k == 0)        ? c : c - xy;
    int t = (k == nz-1)     ? c : c + xy;
    f2[c] = cc * f1[c] + cw * f1[w] + ce * f1[e] + cs * f1[s]
        + cn * f1[n] + cb * f1[b] + ct * f1[t];
    c += bdimx;
    w = c - 1;
    e = (i+bdimx == nx-1)     ? c : c + 1;
    n += bdimx;
    s += bdimx;
    b += bdimx;
    t += bdimx;
    f2[c] = cc * f1[c] + cw * f1[w] + ce * f1[e] + cs * f1[s]
        + cn * f1[n] + cb * f1[b] + ct * f1[t];
    c += xy-bdimx;
  }
  return;
}
#else
__global__ void diffusion_kernel_xy(REAL *f1, REAL *f2,
                                    int nx, int ny, int nz,
                                    REAL ce, REAL cw, REAL cn, REAL cs,
                                    REAL ct, REAL cb, REAL cc) {
  int bdimx = blockDim.x;  
  int i = blockDim.x * blockIdx.x *2 + threadIdx.x;  
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  int c = i + j * nx;
  int xy = nx * ny;
  int w = (i == 0)        ? c : c - 1;
  int e = (i == nx-1)     ? c : c + 1;
  int n = (j == 0)        ? c : c - nx;
  int s = (j == ny-1)     ? c : c + nx;
  REAL t1, t2, t3;
  REAL t1_2, t2_2, t3_2;  
  t1 = t2 = f1[c];
  t3 = f1[c+xy];
  f2[c] = cc * t2 + cw * f1[w] + ce * f1[e] + cs * f1[s]
      + cn * f1[n] + cb * t1 + ct * t3;
  //
  c += bdimx;
  int w2 = c - 1;
  int e2 = ((i+bdimx) == nx-1)     ? c : c + 1;
  n += bdimx;
  s += bdimx;
  t1_2 = t2_2 = f1[c];
  t3_2 = f1[c+xy];
  f2[c] = cc * t2_2 + cw * f1[w2] + ce * f1[e2] + cs * f1[s]
      + cn * f1[n] + cb * t1_2 + ct * t3_2;
  //
  c += xy-bdimx;
  w += xy; w2 += xy;
  e += xy; e2 += xy;
  n += xy-bdimx;
  s += xy-bdimx;
#pragma unroll 8
  for (int k = 1; k < nz-1; ++k) {
    t1 = t2;
    t2 = t3;
    t3 = f1[c+xy];
    f2[c] = cc * t2 + cw * f1[w] + ce * f1[e] + cs * f1[s]
        + cn * f1[n] + cb * t1 + ct * t3;
    c += bdimx;
    w += xy;
    e += xy; 
    n += bdimx;
    s += bdimx;
    t1_2 = t2_2;
    t2_2 = t3_2;
    t3_2 = f1[c+xy];
    f2[c] = cc * t2_2 + cw * f1[w2] + ce * f1[e2] + cs * f1[s]
        + cn * f1[n] + cb * t1_2 + ct * t3_2;
    c += xy-bdimx;
    w2 += xy;
    e2 += xy;
    n += xy-bdimx;
    s += xy-bdimx;
  }
  t1 = t2;
  t2 = t3;
  f2[c] = cc * t2 + cw * f1[w] + ce * f1[e] + cs * f1[s]
      + cn * f1[n] + cb * t1 + ct * t3;
  c += bdimx;
  n += bdimx;
  s += bdimx;
  t1_2 = t2_2;
  t2_2 = t3_2;
  f2[c] = cc * t2_2 + cw * f1[w2] + ce * f1[e2] + cs * f1[s]
      + cn * f1[n] + cb * t1_2 + ct * t3_2;
  return;
}
#endif

void Diffusion3DCUDA::InitializeBenchmark() {
  size_t s = sizeof(REAL) * nx_ * ny_ * nz_;
  CUDA_SAFE_CALL(hipHostMalloc((void**)&f1_, s));
  Initialize(f1_, nx_, ny_, nz_,
             kx_, ky_, kz_, dx_, dy_, dz_,
             kappa_, 0.0);
  CUDA_SAFE_CALL(hipMalloc((void**)&f1_d_, s));
  CUDA_SAFE_CALL(hipMalloc((void**)&f2_d_, s));
  CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>(diffusion_kernel),
                                        hipFuncCachePreferL1));
  CUDA_SAFE_CALL(hipEventCreate(&ev1_));
  CUDA_SAFE_CALL(hipEventCreate(&ev2_));
}

void Diffusion3DCUDA::FinalizeBenchmark() {
  assert(f1_);
  CUDA_SAFE_CALL(hipHostFree(f1_));
  assert(f1_d_);
  CUDA_SAFE_CALL(hipFree(f1_d_));
  assert(f2_d_);
  CUDA_SAFE_CALL(hipFree(f2_d_));
}


void Diffusion3DCUDA::RunKernel(int count) {
  size_t s = sizeof(REAL) * nx_ * ny_ * nz_;  
  CUDA_SAFE_CALL(hipMemcpy(f1_d_, f1_, s, hipMemcpyHostToDevice));

  dim3 block_dim(block_x_, block_y_, block_z_);
  dim3 grid_dim(nx_ / block_x_, ny_ / block_y_, 1);

  CUDA_SAFE_CALL(hipEventRecord(ev1_));
  for (int i = 0; i < count; ++i) {
    diffusion_kernel<<<grid_dim, block_dim>>>
        (f1_d_, f2_d_, nx_, ny_, nz_, ce_, cw_, cn_, cs_, ct_, cb_, cc_);
    REAL *t = f1_d_;
    f1_d_ = f2_d_;
    f2_d_ = t;
  }
  CUDA_SAFE_CALL(hipEventRecord(ev2_));
  CUDA_SAFE_CALL(hipMemcpy(f1_, f1_d_, s, hipMemcpyDeviceToHost));
  return;
}

void Diffusion3DCUDA::DisplayResult(int count, float time) {
  Baseline::DisplayResult(count, time);
  float time_wo_pci;
  hipEventElapsedTime(&time_wo_pci, ev1_, ev2_);
  time_wo_pci *= 1.0e-03;
  printf("Kernel-only performance:\n");
  printf("Elapsed time : %.3f (s)\n", time_wo_pci);
  printf("FLOPS        : %.3f (GFLOPS)\n",
         GetGFLOPS(count, time_wo_pci));
  printf("Throughput   : %.3f (GB/s)\n",
         GetThroughput(count ,time_wo_pci));
}

void Diffusion3DCUDAOpt1::InitializeBenchmark() {
  Diffusion3DCUDA::InitializeBenchmark();
  CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>(diffusion_kernel_opt1),
                                        hipFuncCachePreferL1));
}
void Diffusion3DCUDAOpt1::RunKernel(int count) {
  size_t s = sizeof(REAL) * nx_ * ny_ * nz_;  
  CUDA_SAFE_CALL(hipMemcpy(f1_d_, f1_, s, hipMemcpyHostToDevice));

  dim3 block_dim(block_x_, block_y_, block_z_);
  dim3 grid_dim(nx_ / block_x_, ny_ / block_y_, 1);

  CUDA_SAFE_CALL(hipEventRecord(ev1_));
  for (int i = 0; i < count; ++i) {
    diffusion_kernel_opt1<<<grid_dim, block_dim>>>
        (f1_d_, f2_d_, nx_, ny_, nz_, ce_, cw_, cn_, cs_, ct_, cb_, cc_);
    REAL *t = f1_d_;
    f1_d_ = f2_d_;
    f2_d_ = t;
  }
  CUDA_SAFE_CALL(hipEventRecord(ev2_));
  CUDA_SAFE_CALL(hipMemcpy(f1_, f1_d_, s, hipMemcpyDeviceToHost));
  return;
}

void Diffusion3DCUDAOpt2::InitializeBenchmark() {
  Diffusion3DCUDA::InitializeBenchmark();
  CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>(diffusion_kernel_opt2),
                                        hipFuncCachePreferL1));
}
void Diffusion3DCUDAOpt2::RunKernel(int count) {
  size_t s = sizeof(REAL) * nx_ * ny_ * nz_;  
  CUDA_SAFE_CALL(hipMemcpy(f1_d_, f1_, s, hipMemcpyHostToDevice));

  dim3 block_dim(block_x_, block_y_, block_z_);
  dim3 grid_dim(nx_ / block_x_, ny_ / block_y_ / 2, 1);

  CUDA_SAFE_CALL(hipEventRecord(ev1_));
  for (int i = 0; i < count; ++i) {
    diffusion_kernel_opt2<<<grid_dim, block_dim>>>
        (f1_d_, f2_d_, nx_, ny_, nz_, ce_, cw_, cn_, cs_, ct_, cb_, cc_);
    REAL *t = f1_d_;
    f1_d_ = f2_d_;
    f2_d_ = t;
  }
  CUDA_SAFE_CALL(hipEventRecord(ev2_));
  CUDA_SAFE_CALL(hipMemcpy(f1_, f1_d_, s, hipMemcpyDeviceToHost));
  return;
}

void Diffusion3DCUDAXY::InitializeBenchmark() {
  Diffusion3DCUDA::InitializeBenchmark();
  CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>(diffusion_kernel_xy),
                                        hipFuncCachePreferL1));
}

void Diffusion3DCUDAXY::RunKernel(int count) {
  size_t s = sizeof(REAL) * nx_ * ny_ * nz_;  
  CUDA_SAFE_CALL(hipMemcpy(f1_d_, f1_, s, hipMemcpyHostToDevice));

  dim3 block_dim(block_x_, block_y_);
  dim3 grid_dim(nx_ / (block_x_ * 2), ny_ / block_y_, 1);

  CUDA_SAFE_CALL(hipEventRecord(ev1_));
  for (int i = 0; i < count; ++i) {
    diffusion_kernel_xy<<<grid_dim, block_dim>>>
        (f1_d_, f2_d_, nx_, ny_, nz_, ce_, cw_, cn_, cs_, ct_, cb_, cc_);
    REAL *t = f1_d_;
    f1_d_ = f2_d_;
    f2_d_ = t;
  }
  CUDA_SAFE_CALL(hipEventRecord(ev2_));
  CUDA_SAFE_CALL(hipMemcpy(f1_, f1_d_, s, hipMemcpyDeviceToHost));
  return;
}

}

